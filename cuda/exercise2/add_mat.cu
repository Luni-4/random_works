/*
 ============================================================================
 Name        : Esercizio3.cu
 Author      : 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024 // vector size
#define TxB 32 // threads x block

/*
* kernel: somma di vettori
*/
 /*__global__ void add_vect(int *a, int *b, int *c) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
		c[idx] = a[idx] + b[idx];
}*/

__global__ void add_mat(int *a, int *b, int *c) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	int iy = blockDim.y * blockIdx.y + threadIdx.y;
	int idx = iy * gridDim.x + ix;
	if (ix < gridDim.x || iy < gridDim.y)
 		c[idx] = a[idx] + b[idx];
 }


int main(void) {
	const int dimx1 = 80;
	const int dimy1 = 64;
	const int dimx2 = 76;
	const int dimy2 = 62;

	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	int nBytes1 = (dimx1 * dimy1) * sizeof(int);
	int nBytes2 = (dimx2 * dimy2) * sizeof(int);


	// malloc host memory
	a = (int *) malloc(nBytes1);
	b = (int *) malloc(nBytes2);
	c = (int *) malloc(nBytes2);

	// malloc device memory
	hipMalloc((void**) &dev_a, nBytes1);
	hipMalloc((void**) &dev_b, nBytes2);
	hipMalloc((void**) &dev_c, nBytes2);

	// fill the arrays 'a' and 'b' on the CPU
	memset(a, 1, nBytes1);
	memset(b, 1, nBytes2);
	memset(c, 1, nBytes2);

	// copy the arrays 'a' and 'b' to the GPU
	hipMemcpy(dev_a, a, nBytes1, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, nBytes2, hipMemcpyHostToDevice);

	dim3 dimBlock(16,16,1);
	dim3 dimGrid(5,4,1);

	add_mat<<<dimBlock, dimGrid>>>(dev_a, dev_b, dev_c);
	// copy the array 'c' back from the GPU to the CPU
	hipMemcpy(c, dev_c, nBytes2, hipMemcpyDeviceToHost);
	// display the results
	for (int i = 0; i < (dimx2*dimy2); i++) {
		printf("%d\n", c[i]);
	}
	// Free host memory
	free(a);
	free(b);
	free(c);
	// free the memory allocated on the GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;

}

