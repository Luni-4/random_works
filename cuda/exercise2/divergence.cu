#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024 // vector size
#define TxB 32 // threads x block

/*
* kernel: branch
*/


__global__ void pari_dispari_1(int *c) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int a,b;
	a = b = 0;

	if (tid % 2 == 0)
		a = 2;
	else
		b = 1;
	c[tid] = a + b;
}



int main(void) {
	int *c;
	int *dev_c;
	int	nBytes = N * sizeof(int);

	// malloc host memory
	c = (int *)malloc(nBytes);

	// malloc device memory
	hipMalloc((void**) &dev_c, nBytes);

	pari_dispari_1<<<N, TxB>>>(dev_c);

	// copy the array 'c' back from the GPU to the CPU
	hipMemcpy(c, dev_c, nBytes, hipMemcpyDeviceToHost);

	// display the results
	for (int i = 0; i < N; i++) {
		printf("%d\n", c[i]);
	}

	// Free host memory
	free(c);
	// free the memory allocated on the GPU
	hipFree(dev_c);

	return 0;

}
