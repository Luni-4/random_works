/*
 ============================================================================
 Name        : Esercizio2.cu Author      :
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
/*
* Mostra DIMs e IDs di grid, block e thread
*/
__global__ void checkIndex(void) {
	if ((threadIdx.x + threadIdx.y) % 5 == 0) {
	printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) "
			"blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n",
			threadIdx.x, threadIdx.y, threadIdx.z,
			blockIdx.x, blockIdx.y, blockIdx.z,
			blockDim.x, blockDim.y, blockDim.z,
			gridDim.x,gridDim.y,gridDim.z);
	}
}

int main(int argc, char **argv) {
	// definisce grid e struttura dei blocchi
	dim3 block(8, 7, 1);
	dim3 grid(2, 2, 1);
	// controlla dim. dal lato host
	printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
	printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
	// controlla dim. dal lato device
	checkIndex<<<grid, block>>>();
	// reset device
	hipDeviceReset();
	return(0);
}
