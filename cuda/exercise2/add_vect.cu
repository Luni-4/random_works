/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024 // vector size
#define TxB 32 // threads x block

/*
* kernel: somma di vettori
*/
__global__ void add_vect(int *a, int *b, int *c) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
		c[idx] = a[idx] + b[idx];
}


int main(void) {
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	int nBytes = N * sizeof(int);


	// malloc host memory
	a = (int *) malloc(nBytes);
	b = (int *) malloc(nBytes);
	c = (int *) malloc(nBytes);

	// malloc device memory
	hipMalloc((void**) &dev_a, nBytes);
	hipMalloc((void**) &dev_b, nBytes);
	hipMalloc((void**) &dev_c, nBytes);

	// fill the arrays 'a' and 'b' on the CPU
	for	(int i= 0; i< N; i++) {
		a[i] =	rand() % 10;
		b[i] = 	rand() % 10;
	}

	// copy the arrays 'a' and 'b' to the GPU
	hipMemcpy(dev_a, a, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, nBytes, hipMemcpyHostToDevice);


	add_vect<<<N /TxB, TxB>>>(dev_a, dev_b, dev_c);

	// copy the array 'c' back from the GPU to the CPU
	hipMemcpy(c, dev_c, nBytes, hipMemcpyDeviceToHost);

	// display the results
	for (int i = 0; i < N; i++) {
		printf("%d\n", c[i]);
	}

	// Free host memory
	free(a);
	free(b);
	free(c);
	// free the memory allocated on the GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;

}
