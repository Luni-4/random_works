#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define M_PI 3.14159265358979323846


__global__ void tabular(float *a) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	//if (i < n) {
		float x = a[i];
		float s = sinf(x);
		float c = cosf(x);
		a[i] = sqrtf(abs(s * s - c * c));
	//}
}

int main(int argc, char **argv) {
	hipEvent_t start, stop;

	float *h_a, *hp_a;
	float *d_a;
	int N = 1<<15;
	int M = 1<<12;
	int blockSize = 1024;

	// start the timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int nStreams = N / M;

	int nBytes =  N * sizeof(float);
	int bytesPerStream = M * sizeof(float);

	// malloc host memory
	h_a = (float *) malloc(nBytes);

	// malloc device memory
	//hipMalloc((void**) &d_a, nBytes);

	CHECK( hipHostMalloc((void**)&hp_a, nBytes) ); // host pinned
	CHECK( hipMalloc((void**)&d_a, nBytes) );  // device

	float step = M_PI / N;

	for(int i = 0; i < N; i++){
		h_a[i] = i * step;
	}

	// creazione degli stream asincroni non-NULL
	hipStream_t streams[nStreams];

	for (int i = 0; i < nStreams; ++i) {
		CHECK(hipStreamCreate(&streams[i]));
	}

	hipEventRecord(start, 0);

	double iStart, iElaps;
	iStart = seconds();

	for (int i = 0; i < nStreams; i++) {
		int offset = i * bytesPerStream;
		hipMemcpyAsync(&d_a[offset], &h_a[offset], bytesPerStream, hipMemcpyHostToDevice, streams[i]);
		tabular<<<M / blockSize, blockSize, 0, streams[i]>>>(d_a);
		hipMemcpyAsync(&h_a[offset], &d_a[offset], bytesPerStream, hipMemcpyHostToDevice, streams[i]);
	}

	for (int i = 0; i < nStreams; i++) {
		hipStreamSynchronize(streams[i]);
	}

	iElaps = seconds() - iStart;

	// Registro i tempi
	hipEventRecord(stop);

	// Sincronizzo i tempi
	hipEventSynchronize(stop);

	// cleaup finale
	for (int i = 0; i < nStreams; ++i)
		CHECK(hipStreamDestroy(streams[i]));

	// Stampa a
	for(int i = 0; i < N; i++){
			printf("%f\n",h_a[i]);
	}

	// calculate the elapsed time between two events
	float time;
	hipEventElapsedTime(&time, start, stop);

	printf("Tempo CPU: %f\n", iElaps);
	printf("Tempo Gpu: %f\n",time);

	// free memories both host and device
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(h_a);
	hipHostFree(hp_a);
	CHECK(hipFree(d_a));

	// reset device
	CHECK(hipDeviceReset());
	return EXIT_SUCCESS;
}
