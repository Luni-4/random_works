#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 32
#define TxB 32

__global__ void scan_GPU(float *x, float *y){
	__shared__ float smem[TxB];
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < N)
		smem[threadIdx.x] = x[tid];

	// albero di riduzione: scan iterativo
	for (unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
		__syncthreads();
		smem[threadIdx.x] += smem[threadIdx.x - stride];
	}

	y[tid] = smem[threadIdx.x];
}

int main(int argc, char **argv) {
	hipEvent_t start, stop;

	float *h_a, *h_b;
	float *d_a, *d_b;

	// start the timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int nBytes = N  * sizeof(float);

	// malloc host memory
	h_a = (float *) malloc(nBytes);
	h_b = (float *) malloc(nBytes);

	// fill the arrays 'a' on the CPU
	for	(int i= 0; i< N; i++) {
			h_a[i] = i;
	}

	CHECK(hipMalloc((void**)&d_a, nBytes));  // device
	CHECK(hipMalloc((void**)&d_b, nBytes));  // device

	hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);

	scan_GPU<<<N/TxB, TxB>>>(d_a, d_b);

	hipDeviceSynchronize();

	hipMemcpy(h_b, d_b, nBytes, hipMemcpyDeviceToHost);

	// Registro i tempi
	hipEventRecord(stop);

	// Sincronizzo i tempi
	hipEventSynchronize(stop);

	for	(int i= 0; i< N; i++) {
			printf("%.0f ",h_b[i]);
	}

	// calculate the elapsed time between two events
	float time;
	hipEventElapsedTime(&time, start, stop);

	printf("Tempo Gpu: %f\n",time);

	// free memories both host and device
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(h_a);
	free(h_b);
	CHECK(hipFree(d_a));
	CHECK(hipFree(d_b));

	// reset device
	CHECK(hipDeviceReset());
	return EXIT_SUCCESS;
}
