#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "common/common.h"

#define TRIALS_PER_THREAD 10000
#define BLOCKS  264
#define THREADS 264
#define PI 3.1415926535 // known value of pi

float pi_mc_CPU(long trials) {
	long points_in_circle = 0;
	for (long i = 0; i < trials; i++) {
		float x = rand() / (float) RAND_MAX;
		float y = rand() / (float) RAND_MAX;
		points_in_circle += (x * x + y * y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}

__global__ void pi_mc_GPU(float *estimate, hiprandState *states) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	hiprand_init(tid, 0, 0, &states[tid]);
	for (int i = 0; i < TRIALS_PER_THREAD; i++) {
		float x = hiprand_uniform(&states[tid]);
		float y = hiprand_uniform(&states[tid]);
		points_in_circle += (x * x + y * y <= 1.0f);
	}
	estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD;
}

int main(int argc, char *argv[]) {

	float host[BLOCKS * THREADS];
	float *dev;

	// events to measure time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// CPU procedure
	double iStart = seconds();
	float pi_cpu = pi_mc_CPU(THREADS * BLOCKS * TRIALS_PER_THREAD);
	double iElaps = seconds() - iStart;
	printf("CPU elapsed time: %.5f (sec)\n", iElaps);
	printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, abs(pi_cpu - PI));

	// GPU procedure
	hiprandState *devStates;
	hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(float));
	hipMalloc((void **) &devStates, BLOCKS * THREADS * sizeof(hiprandState));
	hipEventRecord(start);
	pi_mc_GPU<<<BLOCKS, THREADS>>>(dev, devStates);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float),
			hipMemcpyDeviceToHost);
	float pi_gpu = 0.0;
	for (int i = 0; i < BLOCKS * THREADS; i++)
		pi_gpu += host[i];
	pi_gpu /= (BLOCKS * THREADS);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU elapsed time (hiprand Monte Carlo): %.5f (sec)\n",
			milliseconds / 1000);
	printf("GPU estimate of PI = %f [error of %f ]\n", pi_gpu,
			abs(pi_gpu - PI));

	hipFree(dev);
	hipFree(devStates);
	return EXIT_SUCCESS;
}
